#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <signal.h>
#include <time.h>
#include <sys/time.h>
#include <chrono>
#include <thread>
#include <math.h>
#include <random>
#include <string.h>

static volatile int keep_running = 1;

// Configuration structure for GPU stress test
struct StressConfig {
    float mean_load;        // Mean GPU load percentage (0-100)
    float std_load;         // Standard deviation of GPU load percentage
    float interval;         // Load adjustment interval in seconds
    int duration;          // Test duration in seconds (0 = unlimited)
    bool verbose;          // Enable verbose output
    bool validate_dist;    // Validate distribution before starting
    unsigned int seed;     // Random seed for reproducible results
};

void signal_handler(int sig) {
    keep_running = 0;
    printf("\nStopping GPU stress test...\n");
}

// Print usage information
void print_usage(const char* program_name) {
    printf("Usage: %s [OPTIONS]\n", program_name);
    printf("Normal Distribution GPU SM Load Stressor\n\n");
    printf("Options:\n");
    printf("  -m, --mean FLOAT     Mean GPU load percentage (0-100, default: 50)\n");
    printf("  -s, --std FLOAT      Standard deviation of GPU load percentage (default: 15)\n");
    printf("  -i, --interval FLOAT Load adjustment interval in seconds (default: 1.0)\n");
    printf("  -t, --time INT       Test duration in seconds (0=unlimited, default: 0)\n");
    printf("  -q, --quiet          Quiet mode - reduce output\n");
    printf("  --validate           Validate normal distribution before starting test\n");
    printf("  --seed INT           Random seed for reproducible results\n");
    printf("  -h, --help           Show this help message\n\n");
    printf("Examples:\n");
    printf("  %s                                    # Use default parameters\n", program_name);
    printf("  %s -m 80 -s 10 -i 1.0               # 80%% mean, 10%% std, 1s interval\n", program_name);
    printf("  %s -m 60 -s 20 -t 300               # Run for 5 minutes\n", program_name);
    printf("  %s --mean 70 --std 15 --quiet       # Quiet mode\n", program_name);
    printf("  %s --validate -m 20 -s 15           # Validate distribution with mean=20, std=15\n", program_name);
    printf("  %s --seed 12345                     # Use specific random seed\n\n", program_name);
    printf("Note:\n");
    printf("  - Load changes follow normal distribution every interval\n");
    printf("  - Values are clamped to [0, 100] range\n");
    printf("  - Use --validate to check distribution statistics\n");
    printf("  - Uses time-slicing method for precise GPU usage control\n");
    printf("  - Press Ctrl+C to stop the test\n");
}

// Parse command line arguments
bool parse_arguments(int argc, char* argv[], StressConfig& config) {
    // Set default values
    config.mean_load = 50.0f;
    config.std_load = 15.0f;
    config.interval = 1.0f;
    config.duration = 0;
    config.verbose = true;
    config.validate_dist = false;
    config.seed = (unsigned int)time(NULL);
    
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-m") == 0 || strcmp(argv[i], "--mean") == 0) {
            if (i + 1 >= argc) {
                printf("Error: Missing value for %s\n", argv[i]);
                return false;
            }
            config.mean_load = atof(argv[++i]);
            if (config.mean_load < 0 || config.mean_load > 100) {
                printf("Error: Mean load must be between 0 and 100\n");
                return false;
            }
        }
        else if (strcmp(argv[i], "-s") == 0 || strcmp(argv[i], "--std") == 0) {
            if (i + 1 >= argc) {
                printf("Error: Missing value for %s\n", argv[i]);
                return false;
            }
            config.std_load = atof(argv[++i]);
            if (config.std_load < 0) {
                printf("Error: Standard deviation must be >= 0\n");
                return false;
            }
        }
        else if (strcmp(argv[i], "-i") == 0 || strcmp(argv[i], "--interval") == 0) {
            if (i + 1 >= argc) {
                printf("Error: Missing value for %s\n", argv[i]);
                return false;
            }
            config.interval = atof(argv[++i]);
            if (config.interval <= 0) {
                printf("Error: Interval must be > 0\n");
                return false;
            }
        }
        else if (strcmp(argv[i], "-t") == 0 || strcmp(argv[i], "--time") == 0) {
            if (i + 1 >= argc) {
                printf("Error: Missing value for %s\n", argv[i]);
                return false;
            }
            config.duration = atoi(argv[++i]);
            if (config.duration < 0) {
                printf("Error: Test duration must be >= 0\n");
                return false;
            }
        }
        else if (strcmp(argv[i], "-q") == 0 || strcmp(argv[i], "--quiet") == 0) {
            config.verbose = false;
        }
        else if (strcmp(argv[i], "--validate") == 0) {
            config.validate_dist = true;
        }
        else if (strcmp(argv[i], "--seed") == 0) {
            if (i + 1 >= argc) {
                printf("Error: Missing value for %s\n", argv[i]);
                return false;
            }
            config.seed = (unsigned int)atoi(argv[++i]);
        }
        else if (strcmp(argv[i], "-h") == 0 || strcmp(argv[i], "--help") == 0) {
            print_usage(argv[0]);
            exit(0);
        }
        else {
            printf("Error: Unknown option %s\n", argv[i]);
            print_usage(argv[0]);
            return false;
        }
    }
    
    return true;
}

// Generate normally distributed load value using Box-Muller transform
float generate_normal_load(float mean, float std_dev, std::mt19937& rng) {
    // Use C++ standard library for more reliable normal distribution
    static std::normal_distribution<float> normal_dist(0.0f, 1.0f);
    
    // Generate standard normal value (mean=0, std=1)
    float z = normal_dist(rng);
    
    // Transform to desired mean and std_dev
    float load = mean + std_dev * z;
    
    // Clamp to valid range [0, 100]
    if (load < 0.0f) load = 0.0f;
    if (load > 100.0f) load = 100.0f;
    
    return load;
}

// Statistical validation function - for debugging
void print_distribution_stats(float mean, float std_dev, int samples = 1000) {
    std::mt19937 rng(12345); // Fixed seed for reproducible stats
    float sum = 0.0f;
    float sum_sq = 0.0f;
    float min_val = 1000.0f;
    float max_val = -1000.0f;
    int clamped_low = 0;
    int clamped_high = 0;
    
    for (int i = 0; i < samples; i++) {
        float val = generate_normal_load(mean, std_dev, rng);
        sum += val;
        sum_sq += val * val;
        
        if (val < min_val) min_val = val;
        if (val > max_val) max_val = val;
        if (val <= 0.0f) clamped_low++;
        if (val >= 100.0f) clamped_high++;
    }
    
    float actual_mean = sum / samples;
    float actual_std = sqrtf((sum_sq / samples) - (actual_mean * actual_mean));
    
    printf("Distribution validation (%d samples):\n", samples);
    printf("  Target: Mean=%.1f%%, StdDev=%.1f%%\n", mean, std_dev);
    printf("  Actual: Mean=%.1f%%, StdDev=%.1f%%\n", actual_mean, actual_std);
    printf("  Range: [%.1f%%, %.1f%%]\n", min_val, max_val);
    printf("  Clamped: %d low (%.1f%%), %d high (%.1f%%)\n", 
           clamped_low, clamped_low*100.0f/samples, 
           clamped_high, clamped_high*100.0f/samples);
    printf("  Theoretical 3-sigma range: [%.1f%%, %.1f%%]\n", 
           mean - 3*std_dev, mean + 3*std_dev);
    printf("\n");
}

// GPU workload kernel - enhanced version to fully utilize SMs
__global__ void gpu_stress_kernel(float *data, int size, int iterations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Ensure each thread has sufficient work
    for (int i = idx; i < size; i += stride) {
        float temp = data[i];
        
        // Execute intensive floating-point operations
        for (int iter = 0; iter < iterations; iter++) {
            temp = temp * 1.000001f + 0.000001f;
            temp = sqrtf(temp * temp + 1.0f);
            temp = sinf(temp) * cosf(temp);
            temp = expf(temp * 0.00001f);
            temp = logf(temp + 1.0f);
            temp = powf(temp, 1.001f);
        }
        
        data[i] = temp;
    }
}

// Print GPU device information
void print_gpu_info() {
    int device_count;
    hipGetDeviceCount(&device_count);
    
    printf("Detected %d GPU device(s)\n", device_count);
    
    for (int i = 0; i < device_count; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        
        printf("\nGPU %d: %s\n", i, prop.name);
        printf("  SM Count: %d\n", prop.multiProcessorCount);
        printf("  Max Threads/SM: %d\n", prop.maxThreadsPerMultiProcessor);
        printf("  Max Threads/Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Max Blocks/SM: %d\n", prop.maxBlocksPerMultiProcessor);
        printf("  Global Memory: %.1f GB\n", prop.totalGlobalMem / (1024.0*1024.0*1024.0));
        printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    }
}

// Get timestamp in milliseconds
long long get_timestamp_ms() {
    auto now = std::chrono::high_resolution_clock::now();
    auto duration = now.time_since_epoch();
    return std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();
}

// Precise sleep function (milliseconds)
void precise_sleep_ms(int ms) {
    if (ms <= 0) return;
    std::this_thread::sleep_for(std::chrono::milliseconds(ms));
}

// Format timestamp for display
std::string format_timestamp() {
    auto now = std::chrono::system_clock::now();
    auto time_t = std::chrono::system_clock::to_time_t(now);
    auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(
        now.time_since_epoch()) % 1000;
    
    char buffer[32];
    struct tm* tm_info = localtime(&time_t);
    strftime(buffer, sizeof(buffer), "%H:%M:%S", tm_info);
    
    char result[64];
    snprintf(result, sizeof(result), "%s.%03d", buffer, (int)ms.count());
    return std::string(result);
}

int main(int argc, char *argv[]) {
    StressConfig config;
    
    // Parse command line arguments
    if (!parse_arguments(argc, argv, config)) {
        return 1;
    }
    
    // Register signal handlers
    signal(SIGINT, signal_handler);
    signal(SIGTERM, signal_handler);
    
    // Initialize CUDA
    hipSetDevice(0);
    
    if (config.verbose) {
        print_gpu_info();
    }
    
    // Get GPU properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    int sm_count = prop.multiProcessorCount;
    
    if (config.verbose) {
        printf("\n");
        printf("======================================================================\n");
        printf("Normal Distribution GPU SM Load Stressor\n");
        printf("======================================================================\n");
        printf("Target load distribution: Mean=%.1f%%, StdDev=%.1f%%\n", 
               config.mean_load, config.std_load);
        printf("Load adjustment interval: %.1fs\n", config.interval);
        printf("Random seed: %u\n", config.seed);
        if (config.duration > 0) {
            printf("Test duration: %d seconds\n", config.duration);
        } else {
            printf("Test duration: Unlimited\n");
        }
        printf("Using time-slicing method for precise GPU usage control\n");
        printf("Press Ctrl+C to stop the test\n");
        printf("======================================================================\n");
        printf("\n\n");
    }
    
    // Initialize random number generator
    std::mt19937 rng(config.seed);
    
    // Validate distribution if requested
    if (config.validate_dist) {
        if (config.verbose) {
            printf("Validating normal distribution...\n");
        }
        print_distribution_stats(config.mean_load, config.std_load, 10000);
        if (!config.verbose) {
            // If we're just validating, exit after showing stats
            return 0;
        }
    }
    
    // Time-slicing parameters
    int base_cycle_time_ms = 100;  // 100ms per cycle
    
    // Configure kernel parameters - fully utilize GPU
    int threads_per_block = 512;
    int blocks_per_sm = 2;  // 2 blocks per SM
    int total_blocks = sm_count * blocks_per_sm;
    int total_threads = total_blocks * threads_per_block;
    
    if (config.verbose) {
        printf("Kernel configuration:\n");
        printf("  Block count: %d\n", total_blocks);
        printf("  Threads per block: %d\n", threads_per_block);
        printf("  Total threads: %d\n", total_threads);
        printf("  Data size: %d MB\n\n", (int)(total_threads * sizeof(float) / (1024*1024)));
    }
    
    // Allocate GPU memory
    float *d_data;
    size_t data_size = total_threads * sizeof(float);
    hipMalloc(&d_data, data_size);
    
    // Initialize data
    float *h_data = (float*)malloc(data_size);
    for (int i = 0; i < total_threads; i++) {
        h_data[i] = (float)(i % 1000) / 1000.0f;
    }
    hipMemcpy(d_data, h_data, data_size, hipMemcpyHostToDevice);
    
    // Calculate kernel iterations to ensure full GPU load during work time
    int kernel_iterations = 5000;
    
    // Statistics
    long long start_time = get_timestamp_ms();
    int iteration_count = 0;
    float current_load = 0.0f;
    float min_load = 1000.0f;
    float max_load = -1000.0f;
    float sum_load = 0.0f;
    
    if (config.verbose) {
        printf("Starting dynamic load test...\n");
        printf("Expected 3-sigma range: [%.1f%%, %.1f%%] (99.7%% of values)\n", 
               fmaxf(0.0f, config.mean_load - 3*config.std_load), 
               fminf(100.0f, config.mean_load + 3*config.std_load));
        printf("Time        | Iter | Target Load | Work/Idle | Running Stats\n");
        printf("------------|------|-------------|-----------|---------------\n");
    }
    
    while (keep_running) {
        long long iteration_start = get_timestamp_ms();
        
        // Generate new load value
        current_load = generate_normal_load(config.mean_load, config.std_load, rng);
        iteration_count++;
        
        // Update statistics
        if (current_load < min_load) min_load = current_load;
        if (current_load > max_load) max_load = current_load;
        sum_load += current_load;
        float avg_load = sum_load / iteration_count;
        
        // Calculate time-slicing parameters for this iteration
        int work_time_ms = (int)(base_cycle_time_ms * current_load / 100.0f);
        int idle_time_ms = base_cycle_time_ms - work_time_ms;
        
        if (config.verbose) {
            printf("%s | %4d | %7.1f%%   | %3dms/%2dms | avg:%.1f%% [%.1f-%.1f%%]\n", 
                   format_timestamp().c_str(), iteration_count, current_load, 
                   work_time_ms, idle_time_ms, avg_load, min_load, max_load);
        }
        
        // Run for the specified interval
        long long interval_end = iteration_start + (long long)(config.interval * 1000);
        
        while (keep_running && get_timestamp_ms() < interval_end) {
            long long cycle_start = get_timestamp_ms();
            
            // Work phase - launch GPU kernel
            if (work_time_ms > 0) {
                long long work_start = get_timestamp_ms();
                
                // Continuously launch kernels until work time is reached
                while (keep_running && (get_timestamp_ms() - work_start) < work_time_ms) {
                    gpu_stress_kernel<<<total_blocks, threads_per_block>>>(
                        d_data, total_threads, kernel_iterations);
                    hipDeviceSynchronize();
                }
            }
            
            // Idle phase - CPU sleep
            if (idle_time_ms > 0 && keep_running) {
                precise_sleep_ms(idle_time_ms);
            }
            
            // Check if we should exit the interval loop
            if (get_timestamp_ms() >= interval_end) {
                break;
            }
        }
        
        // Check duration limit
        if (config.duration > 0) {
            long long elapsed_seconds = (get_timestamp_ms() - start_time) / 1000;
            if (elapsed_seconds >= config.duration) {
                if (config.verbose) {
                    printf("\nTest duration completed (%d seconds)\n", config.duration);
                }
                break;
            }
        }
    }
    
    // Final statistics
    long long total_elapsed = get_timestamp_ms() - start_time;
    
    if (config.verbose) {
        printf("\n");
        printf("========================================\n");
        printf("=== Test Complete ===\n");
        printf("========================================\n");
        printf("Total runtime: %.1f seconds\n", total_elapsed / 1000.0f);
        printf("Total iterations: %d\n", iteration_count);
        printf("Average iterations per second: %.1f\n", 
               iteration_count * 1000.0f / total_elapsed);
        printf("\nLoad Distribution:\n");
        printf("  Target: Mean=%.1f%%, StdDev=%.1f%%\n", 
               config.mean_load, config.std_load);
        printf("  Actual: Mean=%.1f%%, Range=[%.1f%%, %.1f%%]\n", 
               sum_load / iteration_count, min_load, max_load);
        printf("  Expected 3-sigma range: [%.1f%%, %.1f%%]\n", 
               fmaxf(0.0f, config.mean_load - 3*config.std_load), 
               fminf(100.0f, config.mean_load + 3*config.std_load));
        printf("  Final load setting: %.1f%%\n", current_load);
        printf("\nConfiguration:\n");
        printf("  Load adjustment interval: %.1fs\n", config.interval);
        printf("  Random seed: %u\n", config.seed);
    }
    
    // Cleanup resources
    hipFree(d_data);
    free(h_data);
    
    return 0;
}