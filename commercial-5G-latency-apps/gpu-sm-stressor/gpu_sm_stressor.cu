#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <signal.h>
#include <time.h>
#include <sys/time.h>
#include <chrono>
#include <thread>

static volatile int keep_running = 1;

void signal_handler(int sig) {
    keep_running = 0;
    printf("\nStopping GPU stress test...\n");
}

// GPU workload kernel - enhanced version to fully utilize SMs
__global__ void gpu_stress_kernel(float *data, int size, int iterations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Ensure each thread has sufficient work
    for (int i = idx; i < size; i += stride) {
        float temp = data[i];
        
        // Execute intensive floating-point operations
        for (int iter = 0; iter < iterations; iter++) {
            temp = temp * 1.000001f + 0.000001f;
            temp = sqrtf(temp * temp + 1.0f);
            temp = sinf(temp) * cosf(temp);
            temp = expf(temp * 0.00001f);
            temp = logf(temp + 1.0f);
            temp = powf(temp, 1.001f);
        }
        
        data[i] = temp;
    }
}

// Print GPU device information
void print_gpu_info() {
    int device_count;
    hipGetDeviceCount(&device_count);
    
    printf("Detected %d GPU device(s)\n", device_count);
    
    for (int i = 0; i < device_count; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        
        printf("\nGPU %d: %s\n", i, prop.name);
        printf("  SM Count: %d\n", prop.multiProcessorCount);
        printf("  Max Threads/SM: %d\n", prop.maxThreadsPerMultiProcessor);
        printf("  Max Threads/Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Max Blocks/SM: %d\n", prop.maxBlocksPerMultiProcessor);
        printf("  Global Memory: %.1f GB\n", prop.totalGlobalMem / (1024.0*1024.0*1024.0));
        printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    }
}

// Get timestamp in milliseconds
long long get_timestamp_ms() {
    auto now = std::chrono::high_resolution_clock::now();
    auto duration = now.time_since_epoch();
    return std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();
}

// Precise sleep function (milliseconds)
void precise_sleep_ms(int ms) {
    if (ms <= 0) return;
    std::this_thread::sleep_for(std::chrono::milliseconds(ms));
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <GPU_usage_percentage(0-100)>\n", argv[0]);
        printf("Example: %s 50  # Use 50%% of GPU SM\n", argv[0]);
        return 1;
    }
    
    float target_usage = atof(argv[1]);
    if (target_usage < 0 || target_usage > 100) {
        printf("Error: GPU usage percentage must be between 0-100\n");
        return 1;
    }
    
    // Register signal handlers
    signal(SIGINT, signal_handler);
    signal(SIGTERM, signal_handler);
    
    // Initialize CUDA
    hipSetDevice(0);
    print_gpu_info();
    
    // Get GPU properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    int sm_count = prop.multiProcessorCount;
    
    printf("\nStarting GPU stress test, target usage: %.1f%%\n", target_usage);
    printf("Using time-slicing method for precise GPU usage control\n");
    printf("Press Ctrl+C to stop the test\n\n");
    
    // Calculate time-slicing parameters
    int cycle_time_ms = 100;  // 100ms per cycle
    int work_time_ms = (int)(cycle_time_ms * target_usage / 100.0);
    int idle_time_ms = cycle_time_ms - work_time_ms;
    
    printf("Time-slicing configuration:\n");
    printf("  Cycle time: %d ms\n", cycle_time_ms);
    printf("  Work time: %d ms (%.1f%%)\n", work_time_ms, (float)work_time_ms/cycle_time_ms*100);
    printf("  Idle time: %d ms (%.1f%%)\n", idle_time_ms, (float)idle_time_ms/cycle_time_ms*100);
    printf("\n");
    
    // Configure kernel parameters - fully utilize GPU
    int threads_per_block = 512;
    int blocks_per_sm = 2;  // 2 blocks per SM
    int total_blocks = sm_count * blocks_per_sm;
    int total_threads = total_blocks * threads_per_block;
    
    printf("Kernel configuration:\n");
    printf("  Block count: %d\n", total_blocks);
    printf("  Threads per block: %d\n", threads_per_block);
    printf("  Total threads: %d\n", total_threads);
    printf("  Data size: %d MB\n\n", (int)(total_threads * sizeof(float) / (1024*1024)));
    
    // Allocate GPU memory
    float *d_data;
    size_t data_size = total_threads * sizeof(float);
    hipMalloc(&d_data, data_size);
    
    // Initialize data
    float *h_data = (float*)malloc(data_size);
    for (int i = 0; i < total_threads; i++) {
        h_data[i] = (float)(i % 1000) / 1000.0f;
    }
    hipMemcpy(d_data, h_data, data_size, hipMemcpyHostToDevice);
    
    // Calculate kernel iterations to ensure full GPU load during work time
    int kernel_iterations = 5000;
    
    // Statistics
    long long start_time = get_timestamp_ms();
    long long last_print_time = start_time;
    int cycle_count = 0;
    long long total_work_time = 0;
    long long total_idle_time = 0;
    
    printf("Starting test loop...\n");
    
    while (keep_running) {
        long long cycle_start = get_timestamp_ms();
        
        // Work phase - launch GPU kernel
        if (work_time_ms > 0) {
            long long work_start = get_timestamp_ms();
            
            // Continuously launch kernels until work time is reached
            while (keep_running && (get_timestamp_ms() - work_start) < work_time_ms) {
                gpu_stress_kernel<<<total_blocks, threads_per_block>>>(
                    d_data, total_threads, kernel_iterations);
                hipDeviceSynchronize();
            }
            
            total_work_time += get_timestamp_ms() - work_start;
        }
        
        // Idle phase - CPU sleep
        if (idle_time_ms > 0 && keep_running) {
            long long idle_start = get_timestamp_ms();
            precise_sleep_ms(idle_time_ms);
            total_idle_time += get_timestamp_ms() - idle_start;
        }
        
        cycle_count++;
        
        // Print statistics every second
        long long current_time = get_timestamp_ms();
        if (current_time - last_print_time >= 1000) {
            long long elapsed = current_time - start_time;
            float actual_usage = total_work_time * 100.0f / (total_work_time + total_idle_time);
            float cycles_per_sec = cycle_count * 1000.0f / elapsed;
            
            printf("\rTime: %llds | Cycles: %d | Actual usage: %.1f%% | Target: %.1f%% | Cycles/sec: %.1f", 
                   elapsed/1000, cycle_count, actual_usage, target_usage, cycles_per_sec);
            fflush(stdout);
            
            last_print_time = current_time;
        }
    }
    
    // Final statistics
    long long total_elapsed = get_timestamp_ms() - start_time;
    float final_usage = total_work_time * 100.0f / total_elapsed;
    
    printf("\n\n=== Test Complete ===\n");
    printf("Total runtime: %.1f seconds\n", total_elapsed / 1000.0f);
    printf("Total cycles: %d\n", cycle_count);
    printf("Actual GPU usage: %.1f%%\n", final_usage);
    printf("Target usage: %.1f%%\n", target_usage);
    printf("Error: %.1f%%\n", fabs(final_usage - target_usage));
    
    // Cleanup resources
    hipFree(d_data);
    free(h_data);
    
    return 0;
}